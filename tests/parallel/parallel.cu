/*
 * SSC0742 - Programação Concorrente
 * Professor Paulo Sérgio Lopes de Souza
 * Trabalho Prático 4 - Solução Sequencial para um sistema linear utilizando o método de Jacobi-Richardson
 * Grupo 03
 * Integrantes:
 * -> Adriano Belfort de Sousa ­- 7960706
 * -> Giuliano Barbosa Prado -­ 7961109
 ­* -> Henrique de Almeida Machado da Silveira -­ 7961089
 ­* -> Marcello de Paula Ferreira Costa ­- 7960690
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define DOES_NOT_CONVERGE 1
#define CONVERGE 0

#define DEBUG 0
#define DEBUG_LEVEL_2 0

// Os números que compõem a matriz gerada aleatóriamente terão
// valores entre -1024 e 1024
#define MAXVAL 1024
#define ERROR_TOLERANCE 0.0001
#define null NULL

// variável utilizada para controlar quando o limite de erro foi atingido
// de forma a encerrar o método
 __device__ int reachedErrorTolerance = 0;

// kernel para cálculo do valor absoluto de pontos flutuantes
 __device__ float absolute(float x) {
    
    return x < 0.0 ? -x : x;
}

// kernel utilizado para calcular a normalização das matrizes A e B e gerar os valores iniciais para o vetor X
 __device__ void normalize(float *A, float *currentX, float *B, float *normalizedA, float *normalizedB ,int n) {
    int i, j;

    for(i = 0; i < n; i ++) {
        for(j = 0; j < n; j++) {
            if(i == j) {
                normalizedA[i * n + j] = 0.0;
            }
            else {
                normalizedA[i * n + j] = A[i * n + j] / A[i * n + i];
            }
        }
    }

    for(i = 0; i < n; i++) {
        normalizedB[i] = B[i] / A[i * n + i];
        currentX[i] = normalizedB[i];
    }
}

// kernel utilizado para calcular o erro de uma iteracao
__device__ void getError(float *currentX, float *previousX, int n) {
    float maxRelativeError;
    float currentAbsoluteError;
    float currentRelativeError;
    float currentEntry;
    int i;

    currentAbsoluteError = absolute(currentX[0] - previousX[0]);
    currentEntry = absolute(currentX[0]);
    currentRelativeError = currentAbsoluteError/currentEntry;
    maxRelativeError = currentAbsoluteError;

    for(i = 1; i < n; i++) {
        currentAbsoluteError = absolute(currentX[i] - previousX[i]);
        currentEntry = absolute(currentX[i]);
        currentRelativeError = currentAbsoluteError/currentEntry;
        if (currentRelativeError > maxRelativeError){
            maxRelativeError = currentRelativeError;
        }
    }

    // if (DEBUG) {
    //     printf("getError - maxRelativeError [%f]\n", maxRelativeError);
    // }

    if(maxRelativeError < ERROR_TOLERANCE) {
        reachedErrorTolerance = 1;
    }
}

// kernel que computa os valores de X para a iteracao K + 1 a paritr dos valores obtidos na iteracao K.
 __device__ void computeNewCurrentX(float *currentX, float *previousX, float *normalizedA, float *normalizedB, int n, int myIndex, int range) {
    
    // Cada thread calculara uma das posicoes do vetor X
    int i, j;
    float sum;

    // Os calculos sao efetuados, variando-se apenas as colunas da matriz A
    // e as linhas do vetor X da iteracao K
    for(i = 0; i < range; i++) {
        sum = 0.0;
        for(j = 0; j < n; j++) {
            if((myIndex + i) != j) {
                sum -= normalizedA[(myIndex + i) * n + j] * previousX[j];
            }
        }
        // O resultado final e adicionado do valor da linha correspondente
        // do vetor B e finalmente atribuido ao vetor X.
        sum += normalizedB[myIndex];
        currentX[myIndex] = sum;
    }

    

    // Barreira utilizada para que todos os elementos de X sejam calculados antes
    // de que se avance para a proxima etapa
    __syncthreads();

}

// Cada thread copia a sua posicao do vetor X da iteracao atual para a iteracao anterior
 __device__ void copyCurrentXToPreviousX(float *currentX, float *previousX, int myIndex, int range) {
    
    int i;
    for(i = 0; i < range; i++) {
        previousX[myIndex + i] = currentX[myIndex + i];
    }

}

// kernel principal chamado do host. Aqui e definido o esqueleto da solucao
 __global__ void solveJacobiRichardson(float *A, float *B, float *normalizedA, float *normalizedB, float * currentX, float *previousX, int n) {

    // e calculado o indice de cada thread. Se estiver nos limites da dimensao desejada
    int myIndex = threadIdx.x;
    int numThreads = blockDim.x;
    int quoc = 1;
    
    if(myIndex < n) {

        if(n > numThreads) {
            quoc = n/numThreads;
            quoc = quoc == 0 ? 1 : quoc;
            int rest = n % numThreads;
            
            if(myIndex >= rest) {
                myIndex = n - (numThreads - threadIdx.x) * quoc;
            }
            else {
                quoc+=1;
                myIndex = myIndex * quoc;
            }
            
        }

        // Entao a normalizacao acontece uma vez apenas (so para a thread 0)
        if(myIndex == 0) {
            normalize(A, currentX, B, normalizedA, normalizedB, n);
        }

        // Eh repetido o laco enquanto onivel de erro desejado nao for atingido
        do {

            // Primeiramente, passa-se os valores atuais do vetor X para um vetor representando
            // a iteracao passada
            copyCurrentXToPreviousX(currentX, previousX, myIndex, quoc);

            // Sao calculados os valores da iteracao K+1 do vetor X
            computeNewCurrentX(currentX, previousX, normalizedA, normalizedB, n, myIndex, quoc);

            // A checagem de erro eh feita apenas uma vez
            if(myIndex == 0) {
                getError(currentX, previousX, n);
            }
        } while(reachedErrorTolerance == 0);
        // O laco acima eh repetido enquanto nao for atingido o nivel de erro desejado
    }

 }

// Inicializacao de matrizes e vetores do host
__host__ void initialize(float **A, float **currentX, float **B, int *n, FILE *file) {
    fread(n, sizeof(int), 1, file);

    *A = (float *) malloc((*n) * (*n) * sizeof(float));
    *currentX = (float *) malloc(*n * sizeof(float));
    *B = (float *) malloc(*n * sizeof(float));

}

// Dados para popular vetores e matrizes do host sao lidos do arquivo
__host__ void readDataFromInputFile(float *A, float *B, int n, FILE *inputFile) {
    int i, j;

    for(i = 0; i < n; i ++) {
        for(j = 0; j < n; j++) {
            fread(&A[i * n + j], sizeof(float), 1, inputFile);
        }
    }

    for(i = 0; i < n; i ++) {
        fread(&B[i], sizeof(float), 1, inputFile);
    }
}

// Resultados sao transferidos para arquivo
__host__ void showResults(float *A, float *currentX, float *B, int n, FILE *outputFile) {
    int i;
    float calculatedResult = 0.0;
    int line = rand() % n;
        
    for(i = 0; i < n; i++) {
        fprintf(outputFile, "X[%d] = %f\n", i, currentX[i]);
    }
    
    fprintf(outputFile, "\nEquação aleatória para avaliação de corretude:\n");
    for (i = 0; i < n; i++) {
        fprintf(outputFile, "%2.3f * %2.3f", A[line * n + i], currentX[i]);
        calculatedResult += A[line * n + i] * currentX[i];
        if(i != n-1) {
            fprintf(outputFile, " + ");
        }
        else {
            fprintf(outputFile, " = [%2.3f]\n", calculatedResult);
        }
    }
    fprintf(outputFile, "Valor esperado para o resultado:\n%2.3f\n", B[line]);
    fprintf(outputFile, "Diferença entre resultados:\n%2.3f\n", B[line] - calculatedResult);
}

// Funcao de host auxiliar para imprimir valores. Usada durante depuracao
__host__ void printAll(float *A, float *X, float *B, int n) {
    printf("\nA:\n");
    
    int i, j;
    for(i = 0; i < n; i++) {
        for(j = 0; j < n; j++) {
            printf("%f ", A[i * n + j]);
        }
        printf("\n");
    }
    
    printf("\nX:\n");
    
    for(i = 0; i < n; i++) {
        printf("%f ", X[i]);
    }
    printf("\n");
    
    printf("\nB:\n");
    for(i = 0; i < n; i++) {
        printf("%f ", B[i]);
    }
    printf("\n");
}

// Funcao de host para liberar memoria alocada tanto para host quanto para device
__host__ void cleanUp(float *h_A, float *h_currentX, float *h_B, float *d_A, float *d_currentX, float *d_B, float *d_normalizedA, float *d_previousX, float *d_normalizedB) {
    free(h_A);
    free(h_B);
    free(h_currentX);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_currentX);
    hipFree(d_normalizedA);
    hipFree(d_normalizedB);
    hipFree(d_previousX);
    
}

int main(int argc, const char * argv[]) {

    // Arquivos de entrada e saida
    FILE *inputFile = null;   
    FILE *outputFile = null;

    float *h_A; // Matriz A original
    float *h_currentX; // Vetor X - variáveis - valores da iteração atual
    float *h_B; // Vetor B original
 
    int n; // Ordem da matriz A

    // Vetores e matrizes do device
    float *d_A;
    float *d_currentX;
    float *d_B;
    float *d_previousX; 
    float *d_normalizedA;
    float *d_normalizedB;

    // Variaveis para contagem de tempo transcorrido
    clock_t start, end;
    double cpu_time_used;

    // Arquivos sao abertos
    inputFile = fopen(argv[1],"rb");
    if (inputFile == null) {
        perror("Failed to open file");
        exit(0);
    }

    outputFile = fopen(argv[2],"wt");
    if (outputFile == null) {
        perror("Failed to open file");
        exit(0);
    }


    start = clock();

    // Matrizes e vetores do host sao inicializados e dados sao lidos do arquivo de entrada
    initialize(&h_A, &h_currentX, &h_B, &n, inputFile);
    readDataFromInputFile(h_A, h_B, n, inputFile);

    // vetores e matrizes do device sao alocados
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMalloc(&d_currentX, n * sizeof(float));
    hipMalloc(&d_B, n * sizeof(float));
    hipMalloc(&d_previousX, n * sizeof(float));
    hipMalloc(&d_normalizedA, n * n * sizeof(float));
    hipMalloc(&d_normalizedB, n * sizeof(float));

    // Valores dos vetores e matrizes sao copiados para as versoes do device
    hipMemcpy(d_A,h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B, n * sizeof(float), hipMemcpyHostToDevice);

    // Chamada do kernel principal, com 1 bloco e n threads (n eh a dimensao da matriz)
    solveJacobiRichardson<<<1, n>>>(d_A, d_B, d_normalizedA, d_normalizedB, d_currentX, d_previousX, n);

    // Resultados do device transferidos para o host
    hipMemcpy(h_currentX,d_currentX, n * sizeof(float),hipMemcpyDeviceToHost);

    end = clock();

    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Elapsed time: %fs for dimension %d\n", cpu_time_used, n);

    fprintf(outputFile, "*** Parallel Results ***\n");
    showResults(h_A, h_currentX, h_B, n, outputFile);

    fclose(inputFile);
    fclose(outputFile);
    
    cleanUp(h_A, h_currentX, h_B, d_A, d_currentX, d_B, d_normalizedA, d_previousX, d_normalizedB);

    return 0;
}
